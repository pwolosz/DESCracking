#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdint.h"
#include "BinaryHelper.h"
#include "DESEncoder.h"
#include "iterator"
#include "Helpers.h"

__global__ void decrypt(uint64_t* coded_message, uint64_t* message, int key_size, int dev_block_size, int blocks_x, int message_blocks) {
	int index = threadIdx.x;
	int block = blockIdx.x;
	int dim_pow = 0;
	while (dev_block_size > 1) {
		dev_block_size = dev_block_size / 2;
		dim_pow++;
	}
	while (blocks_x > 1) {
		blocks_x = blocks_x / 2;
		dim_pow++;
	}
	uint64_t *encoded;

	int p = key_size - dim_pow;
	uint64_t val = pow2(p);
	if (index != 0 || block != 0) return;
	printf("%llu-%llu\n", index * val, (index + 1) * val - 1);
	for (int j = 1; j <= MAX_MESSAGE_LENGTH; j++) {
		printf("j=%d\n", j);
		uint64_t *messages = get_messages(j);
		printf("Got messages\n");
		for (uint64_t i = index * val; i <= (index+1) * val - 1; i++) {
			for (int k = 0; k < power(ALPHABET_SIZE, j); k++) {
				printf("i=%llu k=%d\n", i, k);
				encoded = encode(new uint64_t[1]{ messages[k] }, i, 1);
				if (encoded[0] == coded_message[0]) {
					message[0] = messages[k];
					return;
				}
			}
		}
	}
}

int main()
{
	uint64_t *messages = get_messages(1);
	uint64_t *encoded = encode(new uint64_t[1]{ messages[0] }, 222, 1);
	int size = 0;
	char *m = new char[MAX_MESSAGE_LENGTH + 1];
	char *message = new char[MAX_MESSAGE_LENGTH + 1];
	uint64_t key = 1;
	printf("Using 32b key and MAX_MESSAGE_LENGTH=%d\nMessage: ", MAX_MESSAGE_LENGTH);
	scanf("%s", m);

	message = get_message(m, &size);

	printf("Coding message: %s\n", message);
	uint64_t *message_blocks = encode_message(message, size);
	uint64_t *encoded_message = encode(message_blocks, key, 1);
	
	printf("%llu\n", encoded_message[0]);

	int key_size = 32;
	int block_size = 512;
	int blocks = 1;
	int used_device_blocks = 4096;
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 0;
	}

	uint64_t *dev_encoded_message = 0;
	cudaStatus = hipMalloc((void**)&dev_encoded_message, blocks * sizeof(uint64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}

	uint64_t *dev_decoded_message=0, *decoded_message = new uint64_t[blocks];
	cudaStatus = hipMalloc((void**)&dev_decoded_message, blocks * sizeof(uint64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}

	cudaStatus = hipMemcpy(dev_encoded_message, encoded_message, blocks * sizeof(uint64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 0;
	}

	decrypt<<<1, 1>>>(dev_encoded_message, dev_decoded_message, key_size, block_size, 4096, blocks);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching decrypt!\n", cudaStatus);
		return 0;
	}

	cudaStatus = hipMemcpy(decoded_message, dev_decoded_message, blocks * sizeof(uint64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 0;
	}

	printf("------------\n");
	printf("Decoded message numerical: %llu\n", decoded_message[0]);
	printf("Decoded message string: %s\n", int_to_string(decoded_message[0]));
}

