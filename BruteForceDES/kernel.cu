#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdint.h"
#include "BinaryHelper.h"
#include "DESEncoder.h"
#include "iterator"
#include "Helpers.h"
#include <ctime>

__global__ void decrypt(uint64_t coded_message, uint64_t* message, int key_size, int dev_block_size, int blocks_x, int message_blocks, int *is_finised, uint64_t *dev_all_messages) {
	int index = threadIdx.x;
	int block = blockIdx.x;
	int dim_pow = 0;
	int dev_b = dev_block_size;
	int deb_i = blocks_x;
	while (dev_b > 1) {
		dev_b = dev_b / 2;
		dim_pow++;
	}
	while (deb_i > 1) {
		deb_i = deb_i / 2;
		dim_pow++;
	}
	uint64_t encoded;
	int p = key_size - dim_pow;
	uint64_t val = pow2(p);

	for (uint64_t i = index * val + block * val*dev_block_size; i <= (index + 1) * val - 1 + block * val*dev_block_size; i++) {
		for (int j = 0; j < get_messages_count(); j++) {
			if (*is_finised == 1) {
				return;
			}
			encoded = encode(dev_all_messages[j], i);
			if (encoded == coded_message) {
				message[0] = dev_all_messages[j];
				*is_finised = 1;
				return;
			}
		}
	}

	printf("%d - %d finished\n ", block, index);
}

uint64_t *allocate_messages() {
	uint64_t *messages = new uint64_t[get_messages_count()];
	int index = 0;

	for (int i = 0; i < MAX_MESSAGE_LENGTH; i++) {
		int m_count = power(ALPHABET_SIZE, i + 1);
		uint64_t *m = get_messages(i + 1);

		for (int j=0; j < m_count; j++) {
			messages[index + j] = m[j];
		}
		index += m_count;
	}


	return messages;
}

int main()
{
	int size = 0;
	char *m = new char[MAX_MESSAGE_LENGTH + 1];
	char *message = new char[MAX_MESSAGE_LENGTH + 1];
	uint64_t key = 3000;
	clock_t begin = clock();
	printf("Using 32b key and MAX_MESSAGE_LENGTH=%d\nMessage: ", MAX_MESSAGE_LENGTH);
	scanf("%s", m);

	message = get_message(m, &size);

	printf("Coding message: %s\n", message);
	printf("Key: %llu\n", key);
	uint64_t message_block = encode_message(message, size);
	uint64_t encoded_message = encode(message_block, key);

	int key_size = 32;
	int block_size = 512;
	int blocks = 1;
	int used_device_blocks = 4096;
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 0;
	}

	uint64_t *dev_all_messages;
	int *is_finised;
	uint64_t *dev_decoded_message=0, *decoded_message = new uint64_t[blocks];
	cudaStatus = hipMalloc((void**)&dev_decoded_message, blocks * sizeof(uint64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}

	cudaStatus = hipMalloc((void**)&is_finised, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}

	uint64_t *messages = allocate_messages();
	cudaStatus = hipMalloc((void**)&dev_all_messages, get_messages_count() * sizeof(uint64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}

	cudaStatus = hipMemcpy(dev_all_messages, messages, get_messages_count() * sizeof(uint64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 0;
	}
	decrypt<<<used_device_blocks, block_size >>>(encoded_message, dev_decoded_message, key_size, block_size, used_device_blocks, blocks, is_finised, dev_all_messages);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching decrypt!\n", cudaStatus);
		return 0;
	}
	clock_t end = clock();
	cudaStatus = hipMemcpy(decoded_message, dev_decoded_message, blocks * sizeof(uint64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 0;
	}

	printf("------------\n");
	printf("Decoded message numerical: %llu\n", decoded_message[0]);
	printf("Decoded message string: %s\n", int_to_string(decoded_message[0]));
	printf("%llu seconds ellapsed\n", uint64_t(end - begin) / CLOCKS_PER_SEC);
	hipFree(dev_decoded_message);
	hipFree(is_finised);

}

