#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdint.h"
#include "BinaryHelper.h"
#include "DESEncoder.h"
#include "iterator"
#include "Helpers.h"

__global__ void decrypt(uint64_t coded_message, uint64_t* message, int key_size, int dev_block_size, int blocks_x, int message_blocks) {
	int index = threadIdx.x;
	int block = blockIdx.x;
	int dim_pow = 0;
	while (dev_block_size > 1) {
		dev_block_size = dev_block_size / 2;
		dim_pow++;
	}
	while (blocks_x > 1) {
		blocks_x = blocks_x / 2;
		dim_pow++;
	}
	uint64_t encoded;

	int p = key_size - dim_pow;
	uint64_t val = pow2(p);
	if (index != 0 || block != 0) return;
	for (int j = 1; j <= MAX_MESSAGE_LENGTH; j++) {
		uint64_t *messages = get_messages(j);
		for (uint64_t i = index * val; i <= (index+1) * val - 1; i++) {
			printf("j=%llu i=%llu\n",j,i);
			for (int k = 0; k < power(ALPHABET_SIZE, j); k++) {
				encoded = encode(messages[k], i);
				if (encoded == coded_message) {
					message[0] = messages[k];
					return;
				}
			}
		}
	}
}

int main()
{
	int size = 0;
	char *m = new char[MAX_MESSAGE_LENGTH + 1];
	char *message = new char[MAX_MESSAGE_LENGTH + 1];
	uint64_t key = 1;
	printf("Using 32b key and MAX_MESSAGE_LENGTH=%d\nMessage: ", MAX_MESSAGE_LENGTH);
	scanf("%s", m);

	message = get_message(m, &size);

	printf("Coding message: %s\n", message);
	uint64_t message_block = encode_message(message, size);
	uint64_t encoded_message = encode(message_block, key);
	
	printf("%llu\n", encoded_message);

	int key_size = 32;
	int block_size = 512;
	int blocks = 1;
	int used_device_blocks = 4096;
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 0;
	}

	uint64_t *dev_encoded_message = 0;


	uint64_t *dev_decoded_message=0, *decoded_message = new uint64_t[blocks];
	cudaStatus = hipMalloc((void**)&dev_decoded_message, blocks * sizeof(uint64_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		return 0;
	}


	decrypt<<<1, 1>>>(encoded_message, dev_decoded_message, key_size, block_size, used_device_blocks, blocks);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching decrypt!\n", cudaStatus);
		return 0;
	}

	cudaStatus = hipMemcpy(decoded_message, dev_decoded_message, blocks * sizeof(uint64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		return 0;
	}

	printf("------------\n");
	printf("Decoded message numerical: %llu\n", decoded_message[0]);
	printf("Decoded message string: %s\n", int_to_string(decoded_message[0]));
}

